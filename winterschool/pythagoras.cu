
#include <hip/hip_runtime.h>
#include <cstdio>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

__global__ void pythagoras(int *pa, int *pb, int *pc, int *pd) {
  int a = *pa;
  int b = *pb;
  int c = *pc;

  if ((a * a + b * b) == c * c)
    *pd = 1;
  else
    *pd = 0;
}

int main(int argc, char *argv[]) {
  if (argc != 4) {
    printf("Usage: %s <num 1> <num 2> <num 3>\n", argv[0]);
    return 0;
  }

  // TODO
  int *m_a, *m_b, *m_c, *m_d;
  CHECK_CUDA(hipMalloc(&m_a, sizeof(int)));
  CHECK_CUDA(hipMalloc(&m_b, sizeof(int)));
  CHECK_CUDA(hipMalloc(&m_c, sizeof(int)));
  CHECK_CUDA(hipMalloc(&m_d, sizeof(int)));

  int a = atoi(argv[1]);
  int b = atoi(argv[2]);
  int c = atoi(argv[3]);
  int d;
  CHECK_CUDA(hipMemcpy(m_a, &a, sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(m_b, &b, sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(m_c, &c, sizeof(int), hipMemcpyHostToDevice));
  pythagoras <<< 80, 256 >>> (m_a, m_b, m_c, m_d);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipMemcpy(&d, m_d, sizeof(int), hipMemcpyDeviceToHost));
  if (d==1){
    printf("YES\n");
  } else{
    printf("NO\n");
  }

  CHECK_CUDA(hipFree(m_a));
  CHECK_CUDA(hipFree(m_b));
  CHECK_CUDA(hipFree(m_c));
  CHECK_CUDA(hipFree(m_d));

  return 0;
}
