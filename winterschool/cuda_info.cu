
#include <hip/hip_runtime.h>
#include <cstdio>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

int main() {
  // TODO
  int count;
  hipDeviceProp_t props[8];
  hipGetDeviceCount(&count);
  printf("Number of devices: %d\n", count);
  for (int d=0; d<count; ++d){
    hipGetDeviceProperties(&props[d], d);
    printf("\tdevice %d\n", d);
    printf("\t\tname: %s\n", props[0].name);
    printf("\t\tmultiProcessorCount: %d\n", props[0].multiProcessorCount);
    printf("\t\tmaxThreadsPerBlock: %d\n", props[0].maxThreadsPerBlock);
    printf("\t\ttotalGlobalMem: %ld\n", props[0].totalGlobalMem);
    printf("\t\tsharedMemPerBlock: %ld\n", props[0].sharedMemPerBlock);    
  }

  return 0;
}
